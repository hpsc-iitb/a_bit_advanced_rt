#include "hip/hip_runtime.h"
#include <cstdint>
#include <vector>
#include <cstdlib>
#include <stdexcept>
#include <iostream>

#include <flags.hpp>
#include <utils.hpp>
#include <raytrace.hpp>
#include <domainparser.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <chrono>

#ifdef COLOR
    const size_t channels = 3; // rgb
#else
    const size_t channels = 1; // monochrome
#endif

std::vector<FL_TYPE> camera(3, 0);



__global__ 
void primary_rays(float* nodes,float* rays, float* image_plane, float* lights, size_t elem_size)
{
    int _i = blockIdx.x * blockDim.x + threadIdx.x;
    // int _j = get_global_id(1);
    float ax,ay,az,bx,by,bz,cx,cy,cz,e01x,e01y,e01z,e02x,e02y,e02z,\
    tx,ty,tz,u,v,D, Dy, Dz, t, nx, ny, nz, rox, roy, roz, rdx, rdy, rdz, ray_length_1,\
    srox, sroy, sroz, srdx, srdy, srdz, nl_1;

    float min_dis = 1e20;
    int near_ele_num = -1, eid;

    unsigned int sz;//, eidx;

    FL_TYPE lx = lights[0];
    FL_TYPE ly = lights[1];
    FL_TYPE lz = lights[2];

    
    __shared__ float elems_shared[100*9];
    

    rox = rays[_i*6];
    roy = rays[_i*6+1];
    roz = rays[_i*6+2];
        
    rdx = rays[_i*6+3];
    rdy = rays[_i*6+4];
    rdz = rays[_i*6+5];
    
    for(unsigned int _j = 0; _j < elem_size*9; _j+=100*9)
    {
        sz = ((elem_size*9 - _j)>=100*9)?100*9:(elem_size*9 - _j);
        if(threadIdx.x == 0)
        {
            for(unsigned int _k = 0; _k < sz; _k++)
            {
                elems_shared[_k] = nodes[_j + _k];
            }
        }
        __syncthreads();
        for(unsigned int _m = 0; _m < sz; _m+= 9)
        {
            //eidx = _j + _m;
            ax = elems_shared[_m];
            ay = elems_shared[_m+1];
            az = elems_shared[_m+2];

            bx = elems_shared[_m+3];
            by = elems_shared[_m+4];
            bz = elems_shared[_m+5];

            cx = elems_shared[_m+6];
            cy = elems_shared[_m+7];
            cz = elems_shared[_m+8];

            // nx = nodes[_j + 9];
            // ny = nodes[_j + 10];
            // nz = nodes[_j + 11];

            e01x = bx - ax;
            e01y = by - ay;
            e01z = bz - az;
            
            e02x = cx - ax;
            e02y = cy - ay;
            e02z = cz - az;

            nx = e01y * e02z - e01z * e02y;
            ny = e01z * e02x - e01x * e02z;
            nz = e01x * e02y - e01y * e02x;

    //         // e01x = nodes[_j + 12];
    //         // e01y = nodes[_j + 13];
    //         // e01z = nodes[_j + 14];
                    
    //         // e02x = nodes[_j + 15];
    //         // e02y = nodes[_j + 16];
    //         // e02z = nodes[_j + 17];

    //         //nl = nodes[_j + 18];

            D = - (nx * rdx + ny*rdy + nz*rdz);  // |-d e1 e2| = -n.d

            if(fabs(D) < 1e-6)
            {
                continue;
            }

            tx = rox - ax;
            ty = roy - ay;
            tz = roz - az;

            Dy = rdx*(tz*e02y - ty*e02z) + tx*(rdy*e02z - e02y*rdz) + e02x*(ty*rdz - rdy*tz);
            u = Dy / D;

            if(u < 0 || u > 1)
            {
                continue;
            }
            // ts[_i] = elems_shared[_m];

            Dz = rdx*(e01z*ty - e01y*tz) + e01x*(tz*rdy - ty*rdz) + tx*(e01y*rdz - rdy*e01z);

            v = Dz / D;

            if(v < 0 || v + u > 1)
            {
                continue;
            }

            t = (tx*nx + ty*ny + tz*nz)/D;

            if(t < 0)
            {
                continue;
            }
            if(t < min_dis)
            {
                near_ele_num = _j+_m;
                min_dis = t;
            }
        }
        __syncthreads();
    }
    
    // if(min_dis<1e19)
    // {
    //     is_hit[_i] = near_ele_num;
    //     ts[_i] = min_dis;
    // }

    //for (size_t _j = 0; _j < w * h * 6; _j+=6)
    //{

        //for (size_t _j = 0; _j < w * h * 6; _j+=6)
    //{
    if(near_ele_num == -1)
    {
        image_plane[_i] = 0.0;
    }
    else
    {
        eid = near_ele_num;
        ax = nodes[eid];
        ay = nodes[eid+1];
        az = nodes[eid+2];

        bx = nodes[eid+3];
        by = nodes[eid+4];
        bz = nodes[eid+5];

        cx = nodes[eid+6];
        cy = nodes[eid+7];
        cz = nodes[eid+8];

        e01x = bx - ax;
        e01y = by - ay;
        e01z = bz - az;
        
        e02x = cx - ax;
        e02y = cy - ay;
        e02z = cz - az;

        nx = e01y * e02z - e01z * e02y;
        ny = e01z * e02x - e01x * e02z;
        nz = e01x * e02y - e01y * e02x;

        srox = rox + rdx * min_dis;
        sroy = roy + rdy * min_dis;
        sroz = roz + rdz * min_dis;

        srdx = lx - srox;
        srdy = ly - sroy;
        srdz = lz - sroz;
        
        ray_length_1 = 1/sqrt(srdx * srdx + srdy * srdy + srdz * srdz);
        nl_1 = 1/sqrt(nx * nx + ny * ny + nz * nz);
        // srdx *= ray_length_1;
        // srdy *= ray_length_1;
        // srdz *= ray_length_1;

        image_plane[_i] = fabs(srdx * nx + srdy * ny + srdz * nz)*ray_length_1*nl_1;
    }
    // return;  
    __syncthreads();

    for(unsigned int _p = 0; _p < elem_size*9; _p+=100*9)
    {
        sz = ((elem_size*9 - _p)>=100*9)?100*9:(elem_size*9 - _p);
        if(threadIdx.x == 0)
        {
            for(unsigned int _q = 0; _q < sz; _q++)
            {
                elems_shared[_q] = nodes[_p + _q];
            }
        }
        __syncthreads();

        // srox = rox + rdx * min_dis;
        // sroy = roy + rdy * min_dis;
        // sroz = roz + rdz * min_dis;
        
        // rdx = lx - srox;
        // rdy = ly - sroy;
        // rdz = lz - sroz;

        for(unsigned int _r = 0; _r < sz; _r+= 9)
        {
            //eidx = _j + _r;
            if(_p + _r == near_ele_num)
            {
                // image_plane[0] = 0;
                continue;
            }
            ax = elems_shared[_r];
            ay = elems_shared[_r+1];
            az = elems_shared[_r+2];

            bx = elems_shared[_r+3];
            by = elems_shared[_r+4];
            bz = elems_shared[_r+5];

            cx = elems_shared[_r+6];
            cy = elems_shared[_r+7];
            cz = elems_shared[_r+8];

            // nx = nodes[_j + 9];
            // ny = nodes[_j + 10];
            // nz = nodes[_j + 11];

            e01x = bx - ax;
            e01y = by - ay;
            e01z = bz - az;
            
            e02x = cx - ax;
            e02y = cy - ay;
            e02z = cz - az;

            nx = e01y * e02z - e01z * e02y;
            ny = e01z * e02x - e01x * e02z;
            nz = e01x * e02y - e01y * e02x;

            D = - (nx * srdx + ny*srdy + nz*srdz);  // |-d e1 e2| = -n.d

            if(fabs(D) < 1e-6)
            {
                continue;
            }

            tx = srox - ax;
            ty = sroy - ay;
            tz = sroz - az;

            Dy = srdx*(tz*e02y - ty*e02z) + tx*(srdy*e02z - e02y*srdz) + e02x*(ty*srdz - srdy*tz);
            u = Dy / D;

            if(u < 0 || u > 1)
            {
                continue;
            }
            // ts[_i] = elems_shared[_m];

            Dz = srdx*(e01z*ty - e01y*tz) + e01x*(tz*srdy - ty*srdz) + tx*(e01y*srdz - srdy*e01z);

            v = Dz / D;

            if(v < 0 || v + u > 1)
            {
                continue;
            }

            t = (tx*nx + ty*ny + tz*nz)/D;

            if(t < 0)
            {
                continue;
            }
            if(t < min_dis)
            {
                image_plane[_i] = 0.0;
                return;
            }
        }
        __syncthreads(); 
    }
}




int main(int argc, char** argv)
{
    camera[0] = 30;
    camera[1] = 30;
    camera[2] = 60;

    std::vector <FL_TYPE> element_vector;
    unsigned int num_of_nodes = 0;
    unsigned int num_of_elements = 0;
    std::string file = "shadow";
    std::cout << "Parsing GMSH domain \"" << file << "\"\n";
    DomainParser(file,element_vector,num_of_nodes,num_of_elements);
    std::cout << "Parsing finished\n" << "Elements: " << element_vector.size() / element_size << "\n";
    
    // create image plane
    // [r g b  r g b ...]
    FL_TYPE *image_plane = \
        (FL_TYPE *)calloc(w * h * channels, sizeof(FL_TYPE));
    if(!image_plane)
    {
        throw std::runtime_error("can't allocate memory for image plane");
    }
    // create the rays
    // will be updated by void RayTrace::updateRays
    // [origin direction   origin direction...]
    FL_TYPE *rays = (FL_TYPE *)malloc(sizeof(FL_TYPE) * w * h * 2 * 3);
    FL_TYPE *shadow_rays = (FL_TYPE *)malloc(sizeof(FL_TYPE) * w * h * 2 * 3);
    if(!rays)
    {
        throw std::runtime_error("can't allocate memory for rays");
    }
    RayTrace::updateRays(camera, rays);

    FL_TYPE *nodes = &element_vector[0];
    FL_TYPE *ts = (FL_TYPE *)malloc(sizeof(FL_TYPE) * w * h);
    FL_TYPE *is_hit = (FL_TYPE *)malloc(sizeof(FL_TYPE) * w * h);
    for(size_t _i = 0; _i < w*h; _i++)
    {
        ts[_i] = 1e20;
        is_hit[_i] = -1;
    }
    
    FL_TYPE *drays, *dnodes, *dimage_plane, *dlights;//, *dts;
    FL_TYPE lights[] = {lx, ly, lz};

    hipMalloc((void **)&drays, w*h*6*sizeof(FL_TYPE));
    hipMalloc((void **)&dimage_plane, w*h*sizeof(FL_TYPE));
    hipMalloc((void **)&dlights, 3*sizeof(FL_TYPE));
    hipMalloc((void **)&dnodes, element_vector.size()*sizeof(FL_TYPE));

    hipMemcpy(dimage_plane, image_plane, w*h*sizeof(FL_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(dlights, lights, 3*sizeof(FL_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(dnodes, nodes, element_vector.size()*sizeof(FL_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(drays, rays, w*h*6*sizeof(FL_TYPE), hipMemcpyHostToDevice);

    // render(rays, nodes, element_vector.size() / element_size, lights, 1, image_plane);
    //std::cout << "zzz\n";
    auto start_time = std::chrono::high_resolution_clock::now();

    primary_rays<<<h, w>>>(dnodes, drays, dimage_plane, dlights, (element_vector.size()/element_size));

    hipMemcpy(image_plane, dimage_plane, w*h*sizeof(FL_TYPE), hipMemcpyDeviceToHost);

    auto end_time = std::chrono::high_resolution_clock::now();
    double time_spent = std::chrono::duration<double, std::milli>(end_time - start_time).count();
    std::cout << "Render time taken: " << time_spent << "ms\n";
    //hipMemcpy(drays, shadow_rays, w*h*6*sizeof(FL_TYPE), hipMemcpyHostToDevice);
    

    // for(size_t _i = 0; _i < h; _i++)
    // {
    //     for(size_t _j = 0; _j < w; _j++)
    //     {
    //         std::cout << image_plane[_i*w + _j] << " ";
    //     }
    //     std::cout << std::endl;
        
    // }
    

    RayTrace::writeImage(image_plane, "a.ppm");
    hipFree(drays);
    hipFree(dlights);
    hipFree(dimage_plane);
    hipFree(dnodes);
}
