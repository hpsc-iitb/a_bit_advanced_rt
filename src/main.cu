#include "hip/hip_runtime.h"
#include <cstdint>
#include <vector>
#include <cstdlib>
#include <stdexcept>
#include <iostream>

#include <flags.hpp>
#include <utils.hpp>
#include <raytrace.hpp>
#include <domainparser.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <chrono>

#ifdef COLOR
    const size_t channels = 3; // rgb
#else
    const size_t channels = 1; // monochrome
#endif

std::vector<FL_TYPE> camera(3, 0);



__global__ 
void checkintersection(float* nodes,float* rays, float* ts, size_t elem_size)
{
    int _i = blockIdx.x * blockDim.x + threadIdx.x;
    // int _j = get_global_id(1);
    float ax,ay,az,bx,by,bz,cx,cy,cz,e01x,e01y,e01z,e02x,e02y,e02z,\
    tx,ty,tz,u,v,D, Dy, Dz, t, nx, ny, nz, rox, roy, roz, rdx, rdy, rdz;

    float min_dis = 1e20;

    unsigned int sz, eidx;
    
    __shared__ float elems_shared[100*9];
    

    rox = rays[_i*6];
    roy = rays[_i*6+1];
    roz = rays[_i*6+2];
        
    rdx = rays[_i*6+3];
    rdy = rays[_i*6+4];
    rdz = rays[_i*6+5];
    
    for(unsigned int _j = 0; _j < elem_size*9; _j+=100*9)
    {
        sz = ((elem_size*9 - _j)>=100*9)?100*9:(elem_size*9 - _j);
        if(threadIdx.x == 0)
        {
            for(unsigned int _k = 0; _k < sz; _k++)
            {
                elems_shared[_k] = nodes[_j + _k];
            }
        }
        __syncthreads();
        for(unsigned int _m = 0; _m < sz; _m+= 9)
        {
            eidx = _j + _m;
            ax = elems_shared[_m];
            ay = elems_shared[_m+1];
            az = elems_shared[_m+2];

            bx = elems_shared[_m+3];
            by = elems_shared[_m+4];
            bz = elems_shared[_m+5];

            cx = elems_shared[_m+6];
            cy = elems_shared[_m+7];
            cz = elems_shared[_m+8];

            // nx = nodes[_j + 9];
            // ny = nodes[_j + 10];
            // nz = nodes[_j + 11];

            e01x = bx - ax;
            e01y = by - ay;
            e01z = bz - az;
            
            e02x = cx - ax;
            e02y = cy - ay;
            e02z = cz - az;

            nx = e01y * e02z - e01z * e02y;
            ny = e01z * e02x - e01x * e02z;
            nz = e01x * e02y - e01y * e02x;

    //         // e01x = nodes[_j + 12];
    //         // e01y = nodes[_j + 13];
    //         // e01z = nodes[_j + 14];
                    
    //         // e02x = nodes[_j + 15];
    //         // e02y = nodes[_j + 16];
    //         // e02z = nodes[_j + 17];

    //         //nl = nodes[_j + 18];

            D = - (nx * rdx + ny*rdy + nz*rdz);  // |-d e1 e2| = -n.d

            if(fabs(D) < 1e-6)
            {
                continue;
            }

            tx = rox - ax;
            ty = roy - ay;
            tz = roz - az;

            Dy = rdx*(tz*e02y - ty*e02z) + tx*(rdy*e02z - e02y*rdz) + e02x*(ty*rdz - rdy*tz);
            u = Dy / D;

            if(u < 0 || u > 1)
            {
                continue;
            }
            // ts[_i] = elems_shared[_m];

            Dz = rdx*(e01z*ty - e01y*tz) + e01x*(tz*rdy - ty*rdz) + tx*(e01y*rdz - rdy*e01z);

            v = Dz / D;

            if(v < 0 || v + u > 1)
            {
                continue;
            }

            t = (tx*nx + ty*ny + tz*nz)/D;

            if(t < 0)
            {
                continue;
            }
            if(t < min_dis)
            {
                min_dis = t;
            }
        }
        __syncthreads();
    }
    if(min_dis < 1e19)
    {
        ts[_i] = min_dis;
    }
}



int main(int argc, char** argv)
{
    camera[0] = 30;
    camera[1] = 30;
    camera[2] = 60;

    std::vector <FL_TYPE> element_vector;
    unsigned int num_of_nodes = 0;
    unsigned int num_of_elements = 0;
    std::string file = "shadow";
    std::cout << "Parsing GMSH domain \"" << file << "\"\n";
    DomainParser(file,element_vector,num_of_nodes,num_of_elements);
    std::cout << "Parsing finished\n" << "Elements: " << element_vector.size() / element_size << "\n";
    
    // create image plane
    // [r g b  r g b ...]
    FL_TYPE *image_plane = \
        (FL_TYPE *)calloc(w * h * channels, sizeof(FL_TYPE));
    if(!image_plane)
    {
        throw std::runtime_error("can't allocate memory for image plane");
    }
    // create the rays
    // will be updated by void RayTrace::updateRays
    // [origin direction   origin direction...]
    FL_TYPE *rays = (FL_TYPE *)malloc(sizeof(FL_TYPE) * w * h * 2 * 3);
    if(!rays)
    {
        throw std::runtime_error("can't allocate memory for rays");
    }
    RayTrace::updateRays(camera, rays);

    FL_TYPE *nodes = &element_vector[0];
    FL_TYPE *ts = (FL_TYPE *)malloc(sizeof(FL_TYPE) * w * h);
    for(size_t _i = 0; _i < w*h; _i++)
    {
        ts[_i] = 1e20;
    }
    

    FL_TYPE *drays, *dnodes, *dts;

    hipMalloc((void **)&drays, w*h*6*sizeof(FL_TYPE));
    hipMalloc((void **)&dts, w*h*sizeof(FL_TYPE));
    hipMalloc((void **)&dnodes, element_vector.size()*sizeof(FL_TYPE));

    hipMemcpy(dts, ts, w*h*sizeof(FL_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(dnodes, nodes, element_vector.size()*sizeof(FL_TYPE), hipMemcpyHostToDevice);
  
    FL_TYPE lights[] = {lx, ly, lz};
    // render(rays, nodes, element_vector.size() / element_size, lights, 1, image_plane);
    std::cout << "zzz\n";
    auto start_time = std::chrono::high_resolution_clock::now();
    hipMemcpy(drays, rays, w*h*6*sizeof(FL_TYPE), hipMemcpyHostToDevice);

    checkintersection<<<h, w>>>(dnodes, drays, dts, (element_vector.size()/element_size));

    hipMemcpy(ts, dts, w*h*sizeof(FL_TYPE), hipMemcpyDeviceToHost);

    auto end_time = std::chrono::high_resolution_clock::now();
    double time_spent = std::chrono::duration<double, std::milli>(end_time - start_time).count();
    std::cout << "Render time taken: " << time_spent << "ms\n";

    // for(size_t _i = 0; _i < h; _i++)
    // {
    //     for(size_t _j = 0; _j < w; _j++)
    //     {
    //         std::cout << ts[_i*w + _j] << " ";
    //     }
    //     std::cout << std::endl;
        
    // }
    

    // RayTrace::writeImage(image_plane, "a.ppm");
    hipFree(drays);
    hipFree(dts);
    hipFree(dnodes);
}


// TODO: normalize rays