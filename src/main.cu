#include "hip/hip_runtime.h"
#include <cstdint>
#include <vector>
#include <cstdlib>
#include <stdexcept>
#include <iostream>
#include <algorithm>
#include <chrono>

#include <flags.hpp>
#include <utils.hpp>
#include <raytrace.hpp>
#include <domainparser.hpp>
#include <tree.hpp>


#include <SFML/Window.hpp>
#include <SFML/Graphics.hpp>
#include <SFML/System.hpp>


#ifdef COLOR
    const size_t channels = 3; // rgb
#else
    const size_t channels = 1; // monochrome
#endif

std::vector<FL_TYPE> camera(3, 0);
std::vector<FL_TYPE> domain_limits(6, 0);

__device__
void d_normalize(
    float &x, float &y, float &z,
    float &nx, float &ny, float &nz
)
{
    float l = sqrt(x * x + y * y + z * z);
    nx = x / l;
    ny = y / l;
    nz = z / l;
}

__device__
bool d_rayTreeIntersection(
    float rox, float roy, float roz,
    float rdx, float rdy, float rdz,
    float *tree, float *tree_idx, unsigned int *ids,
    unsigned int &idx, unsigned int current_node,
    bool normalized
)
{
    unsigned int ctpos = (unsigned int)tree_idx[current_node];
    if(tree[ctpos + 26] == 0)
    {
        return false;
    }
    rdx = (!rdx)?1e-8:rdx;
    rdy = (!rdy)?1e-8:rdy;
    rdz = (!rdz)?1e-8:rdz;
    
    if(!normalized)
    {
        d_normalize(rdx, rdy, rdz, rdx, rdy, rdz);
    }

    float swap_tmp;

    float txmin = (tree[ctpos+1+0] - rox) / rdx; // vertex a
    float txmax = (tree[ctpos+1+3] - rox) / rdx; // vertex b
    
    float tymin = (tree[ctpos+1+7] - roy) / rdy; // vertex c
    float tymax = (tree[ctpos+1+1] - roy) / rdy; // vertex a

    float tzmin = (tree[ctpos+1+2] - roz) / rdz; // vertex a
    float tzmax = (tree[ctpos+1+14] - roz) / rdz; // vertex e


    // account for negatives
    if(rdx < 0)
    {
        swap_tmp = txmax;
        txmax = txmin;
        txmin = swap_tmp;
    }
    if(rdy < 0)
    {
        swap_tmp = tymax;
        tymax = tymin;
        tymin = swap_tmp;
    }
    if(rdz < 0)
    {
        swap_tmp = tzmax;
        tzmax = tzmin;
        tzmin = swap_tmp;
    }

    float tmin = fmax(
        txmin, fmax(tymin, tzmin)
    );

    float tmax = fmin(
        txmax, fmin(tymax, tzmax)
    );

    if(tmin <= tmax)
    {
        // there exists a parameter t for which ray intersects nodes
        if(tree[ctpos+25])
        {
            // last node, add own id to intersecting nodes
            ids[idx++] = tree[ctpos];
            return true;
        }
        else
        {
            bool retval = false;
            for(unsigned int _k = 0; _k < 8; _k++)
            {
                retval = retval | d_rayTreeIntersection(
                    rox, roy, roz, rdx, rdy, rdz,
                    tree, tree_idx, ids, idx, tree[ctpos+27+_k],
                    true
                );
            }
            return retval;
        }
    }
    else
    {
        return false;
    }
}

__global__
void render_gpu(
    float *rays, float *elems, float *tree, float *tree_idx,
    float *image_plane, float *lights, int num_lights,
    int num_tree_sz, int elem_sz
)
{
    float rox, roy, roz, rdx, rdy, rdz, nrdx, nrdy, nrdz;
    unsigned int pixel_num = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int ray_start = pixel_num * 6;
    image_plane[pixel_num] = 0.0;
    unsigned int ids[20];
    unsigned int idx = 0;

    // unsigned int ct = 0;
    
    unsigned long _j;
    
    float t; // parametric eqn
    // float illum;
    float u, v; // barycentric coords
    float ax, ay, az;//, bx, by, bz, cx, cy, cz; // nodes
    // float px, py, pz; // intersection point
    float nx, ny, nz, nl; // surface normal
    float e01x, e01y, e01z, e02x, e02y, e02z;
    // float ray_length_1; // 1/shadow_ray_length
    float tx, ty, tz, D, Dy, Dz;

    unsigned int hit_elem = -1;

    rox = rays[ray_start];
    roy = rays[ray_start+1];
    roz = rays[ray_start+2];

    rdx = rays[ray_start+3];
    rdy = rays[ray_start+4];
    rdz = rays[ray_start+5];
    // image_plane[pixel_num] = 0.5;
    float min_dis = 1e20;
    unsigned int leaf_pos;
    if(d_rayTreeIntersection(
        rox, roy, roz, rdx, rdy, rdz,
        tree, tree_idx, ids, idx, 0, false
    ))
    {
        for(unsigned int _i = 0; _i < idx; _i++)
        {
            leaf_pos = (unsigned int)tree_idx[ids[_i]];
            for(unsigned int _m = 0; _m < tree[leaf_pos+26]; _m++)
            {
                _j = tree[leaf_pos + 27 + _m] * 19;
                ax = elems[_j];
                ay = elems[_j+1];
                az = elems[_j+2];

                nx = elems[_j + 9];
                ny = elems[_j + 10];
                nz = elems[_j + 11];

                e01x = elems[_j + 12];
                e01y = elems[_j + 13];
                e01z = elems[_j + 14];
                
                e02x = elems[_j + 15];
                e02y = elems[_j + 16];
                e02z = elems[_j + 17];

                nl = elems[_j + 18];

                D = - nl*(nx*rdx + ny*rdy + nz*rdz);  // |-d e1 e2| = -n.d

                if(fabs(D) < 1e-6)
                {
                    continue;
                }

                tx = rox - ax;
                ty = roy - ay;
                tz = roz - az;
                // FL_TYPE px, py, pz;
                // cross(rdx, rdy, rdz, e02x, e02y, e02z, px, py, pz);

                Dy = rdx*(tz*e02y - ty*e02z) + tx*(rdy*e02z - e02y*rdz) + e02x*(ty*rdz - rdy*tz);
                u = Dy / D;

                if(u < 0 || u > 1)
                {
                    continue;
                }

                Dz = rdx*(e01z*ty - e01y*tz) + e01x*(tz*rdy - ty*rdz) + tx*(e01y*rdz - rdy*e01z);

                v = Dz / D;

                if(v < 0 || v + u > 1)
                {
                    continue;
                }

                t = (tx*nx + ty*ny + tz*nz)*nl/D;

                if(t < 0)
                {
                    continue;
                }
                if(t < min_dis)
                {
                    // near_ele_num = _j+_m;
                    hit_elem = _j;
                    min_dis = t;
                }
            }
        }
    }

    if(min_dis < 1e19)
    {
        rox = rox + rdx*min_dis;
        roy = roy + rdy*min_dis;
        roz = roz + rdz*min_dis;

        rdx = lights[0] - rox;
        rdy = lights[1] - roy;
        rdz = lights[2] - roz;

        d_normalize(rdx, rdy, rdz, nrdx, nrdy, nrdz);

        image_plane[pixel_num] = fabs(elems[hit_elem+9]*nrdx + elems[hit_elem+10]*nrdy + elems[hit_elem+11]*nrdz);
    }
    
    idx = 0;
    // return;

    if(d_rayTreeIntersection(
        rox, roy, roz, rdx, rdy, rdz,
        tree, tree_idx, ids, idx, 0, false
    ))
    {
        // return;
    // image_plane[pixel_num] = ((float)idx)/10.0;
        for(unsigned int _i = 0; _i < idx; _i++)
        {
            leaf_pos = (unsigned int)tree_idx[ids[_i]];
            for(unsigned int _m = 0; _m < tree[leaf_pos+26]; _m++)
            {
                _j = tree[leaf_pos + 27 + _m] * 19;
                if(_j == hit_elem)
                {
                    continue;
                }
                ax = elems[_j];
                ay = elems[_j+1];
                az = elems[_j+2];

                // bx = elems[_j+3];
                // by = elems[_j+4];
                // bz = elems[_j+5];

                // cx = elems[_j+6];
                // cy = elems[_j+7];
                // cz = elems[_j+8];

                nx = elems[_j + 9];
                ny = elems[_j + 10];
                nz = elems[_j + 11];

                e01x = elems[_j + 12];
                e01y = elems[_j + 13];
                e01z = elems[_j + 14];
                
                e02x = elems[_j + 15];
                e02y = elems[_j + 16];
                e02z = elems[_j + 17];

                nl = elems[_j + 18];

                D = - nl*(nx * rdx + ny*rdy + nz*rdz);  // |-d e1 e2| = -n.d

                if(fabs(D) < 1e-6)
                {
                    continue;
                }

                tx = rox - ax;
                ty = roy - ay;
                tz = roz - az;
                // FL_TYPE px, py, pz;
                // cross(rdx, rdy, rdz, e02x, e02y, e02z, px, py, pz);

                Dy = rdx*(tz*e02y - ty*e02z) + tx*(rdy*e02z - e02y*rdz) + e02x*(ty*rdz - rdy*tz);
                u = Dy / D;

                if(u < 0 || u > 1)
                {
                    continue;
                }

                Dz = rdx*(e01z*ty - e01y*tz) + e01x*(tz*rdy - ty*rdz) + tx*(e01y*rdz - rdy*e01z);

                v = Dz / D;

                if(v < 0 || v + u > 1)
                {
                    continue;
                }

                t = (tx*nx + ty*ny + tz*nz)*nl/D;

                if(t < 0)
                {
                    continue;
                }
                image_plane[pixel_num] = 0.0;
                return;
            }
        }
    }

}

int main(int argc, char** argv)
{
    camera[0] = 30;
    camera[1] = 30;
    camera[2] = 60;

    std::vector <FL_TYPE> element_vector;
    // unsigned int num_of_nodes;
    // unsigned int num_of_elements;
    std::string file = "shadow";
    std::cout << "Parsing GMSH domain \"" << file << "\"\n";
    DomainParser(file,element_vector, domain_limits);

    std::cout << "Parsing finished\n" << "Elements: " << element_vector.size() / element_size << "\n";
    std::cout << "Domain limits: [" << domain_limits[0] << ", "\
        << domain_limits[1] << ", "\
        << domain_limits[2] << "], ["\
        << domain_limits[3] << ", "\
        << domain_limits[4] << ", "\
        << domain_limits[5] << "]\n";
    
    FL_TYPE max_length = std::max(
        fabs(domain_limits[0] - domain_limits[3]),
        std::max(
            fabs(domain_limits[1] - domain_limits[4]),
            fabs(domain_limits[2] - domain_limits[5])
        )
    );

            
    std::cout << "Domain max length: " << max_length << "\n";

    Node::node_count = 0;
    Node root(
        domain_limits[0] - fabs(max_length*(1-tree_minus_tol)),
        domain_limits[4] + fabs(max_length*(1-tree_plus_tol)),
        domain_limits[2] - fabs(max_length*(1-tree_minus_tol)),
        max_length*tree_plus_tol/tree_minus_tol,
        max_length*tree_plus_tol/(tree_minus_tol*powf(2, max_depth))
    );
        
    std::cout << "Octree nodes: " << Node::node_count << " \n";


    // create image plane
    // [r g b  r g b ...]
    // create the rays
    // will be updated by void RayTrace::updateRays
    // [origin direction   origin direction...]
    FL_TYPE *rays = (FL_TYPE *)malloc(sizeof(FL_TYPE) * w * h * 2 * 3);
    if(!rays)
    {
        throw std::runtime_error("can't allocate memory for rays");
    }

    FL_TYPE *nodes = &element_vector[0];
    
    fillTree(nodes, element_vector.size() / element_size);
    root.numElementsInside();
    std::cout<< "tree has: " << root.numElementsInside() << " elements\n";
  
    FL_TYPE lights[] = {lx, ly, lz};
    
    std::vector<FL_TYPE> vec_tree(0), vec_tree_ids(0);
    flattenTree(vec_tree, vec_tree_ids);

    sf::RenderWindow window(sf::VideoMode(w, h), "Render");    
    
    sf::Uint8 *sf_pixbuf = new sf::Uint8[w*h*4]; // rgba

    FL_TYPE *image_plane = \
    (FL_TYPE *)calloc(w * h * channels, sizeof(FL_TYPE));
    if(!image_plane)
    {
        throw std::runtime_error("can't allocate memory for image plane");
    }


    float *d_tree, *d_tree_idx, *d_elems, *d_rays, *d_image_plane, *d_lights;
    
    hipMalloc((void **)&d_rays, w*h*6*sizeof(FL_TYPE));
    hipMalloc((void **)&d_image_plane, w*h*sizeof(FL_TYPE));
    hipMalloc((void **)&d_lights, 3*sizeof(FL_TYPE));
    hipMalloc((void **)&d_elems, element_vector.size()*sizeof(FL_TYPE));
    hipMalloc((void **)&d_tree, vec_tree.size()*sizeof(FL_TYPE));
    hipMalloc((void **)&d_tree_idx, vec_tree_ids.size()*sizeof(FL_TYPE));
    
    hipMemcpy(d_image_plane, image_plane, w*h*sizeof(FL_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_lights, lights, 3*sizeof(FL_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_elems, nodes, element_vector.size()*sizeof(FL_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_tree, &vec_tree[0], vec_tree.size()*sizeof(FL_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_tree_idx, &vec_tree_ids[0], vec_tree_ids.size()*sizeof(FL_TYPE), hipMemcpyHostToDevice);

    
    while(window.isOpen())
    {
        // image_plane = \
        // (FL_TYPE *)calloc(w * h * channels, sizeof(FL_TYPE));
        // if(!image_plane)
        // {
        //     throw std::runtime_error("can't allocate memory for image plane");
        // }

        auto start_time = std::chrono::high_resolution_clock::now();
        sf::Texture texture;
        texture.create(w, h);
        sf::Sprite sprite(texture);
        

        sf::Event event;
        while(window.pollEvent(event))
        {
            if(event.type == sf::Event::Closed)
            {
                window.close();
            }
            if(event.type == sf::Event::KeyReleased)
            {
                if(event.key.code == sf::Keyboard::Up)
                {
                    camera[2] += 5;
                }
                else if(event.key.code == sf::Keyboard::Down)
                {
                    camera[2] -= 5;
                }
                if(event.key.code == sf::Keyboard::Left)
                {
                    camera[0] -= 0.5;
                }
                else if(event.key.code == sf::Keyboard::Right)
                {
                    camera[0] += 0.5;
                }

                else if(event.key.code == sf::Keyboard::Key::S)
                {
                    RayTrace::writeImage(image_plane, "a.ppm");
                }

            }
            // sf::Image image()
        }

        RayTrace::updateRays(camera, rays);

        // render(rays, nodes, element_vector.size() / element_size, lights, 1, image_plane, root);

        hipMemcpy(d_rays, rays, w*h*6*sizeof(FL_TYPE), hipMemcpyHostToDevice);
        render_gpu<<<w, h>>>(
            d_rays, d_elems, d_tree, d_tree_idx, d_image_plane,
            d_lights, 1, vec_tree_ids.size(), element_vector.size()/element_size
        );
   
        hipMemcpy(image_plane, d_image_plane, w*h*sizeof(FL_TYPE), hipMemcpyDeviceToHost);

        for(size_t _i = 0; _i < w*h; _i++)
        {
            FL_TYPE pp = image_plane[_i];
            pp = (pp > 1.0)?1.0:pp;
            pp = (pp < 0.0)?0.0:pp;
            sf_pixbuf[_i * 4] = sf::Uint8(pp * 255);
            sf_pixbuf[_i * 4 + 1] = sf::Uint8(pp * 255);
            sf_pixbuf[_i * 4 + 2] = sf::Uint8(pp * 255);
            sf_pixbuf[_i * 4 + 3] = (uint8_t) 255;
        }
        texture.update(sf_pixbuf);
        window.draw(sprite);
        window.display();

        auto end_time = std::chrono::high_resolution_clock::now();
        double time_spent = std::chrono::duration<double, std::milli>(end_time - start_time).count();
        std::cout << "Render time taken: " << time_spent << "ms\n";
    }
    free(image_plane);
    free(rays);
    delete sf_pixbuf;
    hipFree(d_rays);
    hipFree(d_image_plane);
    hipFree(d_lights);
    hipFree(d_elems);
    hipFree(d_tree);
    hipFree(d_tree_idx);
}


// TODO: normalize rays